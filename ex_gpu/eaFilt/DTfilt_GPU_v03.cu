#include "hip/hip_runtime.h"
__global__
void transposeKernel(float * __restrict__ inp, float * __restrict__ outp, int inH, int inW) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= inH) 
    return;
  int c = threadIdx.x;
  if(c >= numCh) 
    return;
  //for(int y = 0; y < inH; y++) {
  for(int x = 0; x < inW; x++) {
    //for(int c = 0; c < numCh; c++) {
      outp[(x*inH + y)*numCh + c] = inp[(y*inW + x)*numCh + c];
    //}
  }
  //}
}

void transpose(float *inp, float *outp, int inH, int inW) {
  dim3 threads(numCh, 128);
  dim3 grid(1, (inH-1)/threads.y + 1);
  transposeKernel<<<grid, threads>>>(inp, outp, inH, inW);
}

__global__
void recursiveFiltHKernel(const float * __restrict__ dHoriz, float * __restrict__ imData, int h, int w, float sigma) {
  float a = exp(-sqrt(2.0f) / sigma); // feedback coefficient
  
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w)
    return;
  int c = threadIdx.x;
  if(c >= numCh)
    return;

  //filter top to down
  for(int y = 0; y < h-1; y++) {
    float coef = pow(a,dHoriz[x*(h-1) + y]);
    //for(int c = 0; c < numCh; c++) {
      imData[((y+1)*w + x)*numCh + c] += coef * (imData[(y*w + x)*numCh + c] - imData[((y+1)*w + x)*numCh + c]); 
    //}
  }
  //filter down to top
  for(int y = h-2; y >= 0; y--) {
    float coef = pow(a,dHoriz[x*(h-1) + y]);
    //for(int c = 0; c < numCh; c++) {
      imData[(y*w + x)*numCh + c] += coef * (imData[((y+1)*w + x)*numCh + c] - imData[(y*w + x)*numCh + c]);
    //}
  }
}

void recursiveFiltH(const float * dHoriz, float * imData, int h, int w, float sigma) {
  dim3 threads(numCh, 128); 
  dim3 grid(1, (w-1)/threads.y + 1);
  recursiveFiltHKernel<<<grid, threads>>>(dHoriz, imData, h, w, sigma);
}


GpuMat *distH_d = 0, *distVT_d = 0, *img_d = 0, *imRes_d = 0, *imResT_d = 0;

void computeDomainTransformFiltering(Mat &img, Mat &imRes, float sigma_s, float sigma_r, int numIter) {

  Size s = img.size();
  Mat imgT(img.t());
  Mat distH(s.height, s.width-1, CV_32FC3);
  Mat distVT(s.width, s.height-1, CV_32FC3);

  //allocate memory on GPU
  if(distH_d == 0) {
	//allocate memory on first run
    distH_d = new GpuMat();
    distVT_d = new GpuMat();
    img_d = new GpuMat();
    imRes_d = new GpuMat();
    imResT_d = new GpuMat();
    createContinuous(distH.size(), CV_32FC3, *distH_d);
    createContinuous(distVT.size(), CV_32FC3, *distVT_d);
    createContinuous(img.size(), CV_32FC3, *img_d);
    createContinuous(imRes.size(), CV_32FC3, *imRes_d);
    createContinuous(imRes.t().size(), CV_32FC3, *imResT_d);
  }

  // 1)  generate distance images
  computeDistanceH((float*)distH.data, (float*)img.data, s.height, s.width, sigma_s, sigma_r);
  computeDistanceH((float*)distVT.data, (float*)imgT.data, s.width, s.height, sigma_s, sigma_r);

  //copy data to GPU
  distH_d->upload(distH);
  distVT_d->upload(distVT);
  img_d->upload(img);
  imRes_d->upload(imRes);

  // 2) iterate filter
  for(int i = 0; i < numIter; i++) {
    // determine sigma (eq. 14)
    float sigma_H_i = sigma_s * sqrt(3) * pow(2, numIter - (i + 1)) / sqrt(pow(4, numIter) - 1);

    // filter horizontal
    transpose((float *)imRes_d->data, (float *)imResT_d->data, s.height, s.width);
    recursiveFiltH((float*)distH_d->data, (float *)imResT_d->data, s.width, s.height, sigma_H_i);
    // filter vertical
    transpose((float *)imResT_d->data, (float *)imRes_d->data, s.width, s.height);
    recursiveFiltH((float *)distVT_d->data, (float *)imRes_d->data, s.height, s.width, sigma_H_i);
  }
  hipDeviceSynchronize();
  imRes_d->download(imRes);
}
