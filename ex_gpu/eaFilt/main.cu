#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <unistd.h>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/photo.hpp>
#include <opencv2/cudaarithm.hpp>
//#include <opencv2/gpu/gpu.hpp>
#include "profiling.h"

using namespace std;
using namespace cv;
using namespace cuda; //for opencv3, was 'gpu' in opencv2

const int numCh = 3;
void computeDistanceH(float *dHoriz, float *im, int h, int w, float sigma_s, float sigma_r);
void computeMagnitude(Mat &img, Mat &mag);
int main(int argc, char **argv);

// active implemementation
//#include "DTfilt_CPU.cu" // CPU-only version
//#include "DTfilt_GPU_v01.cu" // GPU baseline version
//#include "DTfilt_GPU_v02.cu" // with texture caches, channels parallelization
#include "DTfilt_GPU_v03.cu" // with improved memory layout
//#include "DTfilt_GPU_v04.cu" // your own ideas...

void computeDistanceH(float *dHoriz, float *im, int h, int w, float sigma_s, float sigma_r) {
  //access pattern of cv::Mat.data is: im[y][x][c] = im[(y*w + x)*numCh + c];

  // domain transform RF (recursive filtering)
  for(int y = 0; y < h; y++) {
    for(int x = 0; x < w-1; x++) {
      // compute L1 norm of horizontal derivative (1 pixel short in x direction)
      float v = 0;
      for(int c = 0; c < numCh; c++) {
        v += fabs(im[(y*w + x+1)*numCh + c] - im[(y*w + x)*numCh + c]);
      }
      // compute derivatives of domain transform
      v = v*sigma_s/sigma_r + 1;
      //v *= 0.01; //for visualization only(!)
      dHoriz[y*(w-1) + x] = v;
      // we do not compute integral, since the recursive filter uses directly the derivatives
    }
  }
}

void computeMagnitude(Mat &img, Mat &mag) {
  int h = img.rows;
  int w = img.cols;

  vector<Mat> planes;
  split(img, planes);

  Mat magXR = Mat(h, w, CV_32FC1);
  Mat magYR = Mat(h, w, CV_32FC1);

  Mat magXG = Mat(h, w, CV_32FC1);
  Mat magYG = Mat(h, w, CV_32FC1);

  Mat magXB = Mat(h, w, CV_32FC1);
  Mat magYB = Mat(h, w, CV_32FC1);

  Sobel(planes[0], magXR, CV_32FC1, 1, 0, numCh);
  Sobel(planes[0], magYR, CV_32FC1, 0, 1, numCh);

  Sobel(planes[1], magXG, CV_32FC1, 1, 0, numCh);
  Sobel(planes[1], magYG, CV_32FC1, 0, 1, numCh);

  Sobel(planes[2], magXB, CV_32FC1, 1, 0, numCh);
  Sobel(planes[2], magYB, CV_32FC1, 0, 1, numCh);

  Mat mag1 = Mat(h,w,CV_32FC1);
  Mat mag2 = Mat(h,w,CV_32FC1);
  Mat mag3 = Mat(h,w,CV_32FC1);

  cv::magnitude(magXR,magYR,mag1);
  cv::magnitude(magXG,magYG,mag2);
  cv::magnitude(magXB,magYB,mag3);

  mag = mag1 + mag2 + mag3;
  mag = 1.0f - mag;
}

int main(int argc, char **argv) {
  
  // parse arguments
  bool useCam = true;
  //bool showOutput = true, showInput = true, showDetailEnh = true, showStylized = true;
  bool showOutput = false, showInput = false, showDetailEnh = false, showStylized = true;
  bool benchmark = false;
  char *filename;
  if (argc < 2) {
    printf("missing argument: must specify mode (cam, image [path], imageNoShow [path], benchmark [path])\n");     
    exit(-1);
  }
  if(strcmp(argv[1],"cam") == 0) {
    useCam = true;
  } else if (strcmp(argv[1], "image") == 0) {
    useCam = false;
    assert(argc == 3);
    filename = argv[2];
  } else if (strcmp(argv[1], "imageNoShow") == 0) {
    useCam = false;
    showOutput = showInput = showDetailEnh = showStylized = false;
    assert(argc == 3);
    filename = argv[2];
  } else if (strcmp(argv[1], "benchmark") == 0) {
    useCam = false;
    showOutput = showInput = showDetailEnh = showStylized = false;
    assert(argc == 3);
    filename = argv[2];
    benchmark = true;
  } else {
    printf("invalid mode specified\n");
    exit(-1);
  }
  
  // get video cam
  VideoCapture *cap;
  if(useCam) {
#ifdef TX1
    cap = new VideoCapture("nvcamerasrc ! video/x-raw(memory:NVMM), width=(int)1280, height=(int)720,format=(string)I420, framerate=(fraction)24/1 ! nvvidconv flip-method=2 ! video/x-raw, format=(string)BGRx ! videoconvert ! video/x-raw, format=(string)BGR ! appsink"); //open the default camera
#else
    cap = new VideoCapture(0);
#endif
  }
  if(useCam && !cap->isOpened()) { // check if we succeeded
    cerr << "Fail to open camera" << endl;
    return -1;
  } else if (useCam) {
    cout << "opened camera" << endl;
  }

  while(true) {

    const float sigma_s = 60;
    const float sigma_r = 0.4;
    const int numIter = 10;

    // get image from camera or file
    Mat img;
    if(useCam) {
      *cap >> img; // get a new frame from camera
    } else {
      img = imread(filename);
    }
    img.convertTo(img, CV_32FC3, 1.0/255);
    Size s = img.size();
    
    // show input image
    if(showInput) {
      imshow("input", img);
    }

    // process image
    Mat imRes = img.clone();
    test_timer_t domainTransformTimer;
    const int numBenchmarkIter = benchmark ? 10 : 0;
    for(int i = 0; i < numBenchmarkIter+1; i++) {
      if(!benchmark || (benchmark && i == 1)) {
	timer_start(&domainTransformTimer);
      }
      computeDomainTransformFiltering(img, imRes, sigma_s, sigma_r, numIter);
    }
    timer_end(&domainTransformTimer);
    if(benchmark) {
      printf("execution time (domain transform filtering): %ld us\n", timer_delta(&domainTransformTimer)/numBenchmarkIter);    
    } else {
      printf("execution time (domain transform filtering): %ld us\n", timer_delta(&domainTransformTimer));
    }

    // show result image
    if(showOutput) {
      imshow("result image", imRes);
    }

    //detail enhancement
    if(showDetailEnh) {
      const float detailEnhancementFactor = 4.0f;
      Mat imDetailEnh((img-imRes)*detailEnhancementFactor + imRes);
      imshow("detail enhancement", imDetailEnh);
    }

    // stylization
    if(showStylized) {
      test_timer_t stylizationTimer;
      timer_start(&stylizationTimer);
      Mat magnitude = Mat(s.height, s.width, CV_32FC1);
      computeMagnitude(imRes,magnitude);
      Mat stylized = Mat(s.height, s.width, CV_32FC3);
      vector <Mat> temp;
      split(imRes,temp);
      for(int c = 0; c < numCh; c++) {
	cv::multiply(temp[c],magnitude,temp[c]);
      }
      merge(temp,stylized);
      timer_end(&stylizationTimer);
      printf("execution time (stylization): %ld us\n", timer_delta(&stylizationTimer));

      test_timer_t stylizationDisplayTimer;
      timer_start(&stylizationDisplayTimer);
      imshow("stylized", stylized);      
      timer_end(&stylizationDisplayTimer);
      printf("display time (stylization): %ld us\n", timer_delta(&stylizationDisplayTimer));

    }

    if(showOutput || showInput || showDetailEnh || showStylized) {
      if(!useCam) {
        waitKey();
        break;
      } else {
	test_timer_t waitKeyTimer;
	timer_start(&waitKeyTimer);
	int key = waitKey(10);
        if(key > 0 && key != 255) 
          break;
	//waitKey(100);
	timer_end(&waitKeyTimer);
	printf("display time (waitKey): %ld us\n", timer_delta(&waitKeyTimer));
      }
    } else {
      break;
    }
  } // end of main loop

  // the camera will be deinitialized automatically in VideoCapture destructor
  if(useCam) {
    cap->release();
    delete cap;
  }

  return 0;
}



