#include "hip/hip_runtime.h"
__global__
void transposeKernel(float *inp, float *outp, int inH, int inW) {
  int y = blockIdx.x*blockDim.x + threadIdx.x;
  if(y >= inH) 
    return;

  for(int x = 0; x < inW; x++) {
    for(int c = 0; c < numCh; c++) {
      outp[(x*inH + y)*numCh + c] = inp[(y*inW + x)*numCh + c];
    }
  }
}

void transpose(float *inp, float *outp, int inH, int inW) {
  dim3 threads(128);
  dim3 grid((inH-1)/threads.x + 1);
  transposeKernel<<<grid, threads>>>(inp, outp, inH, inW);
}

__global__
void recursiveFiltHKernel(const float * dHoriz, float * imData, int h, int w, float sigma) {
	
/* CPU implementation: 
    - recursively performs a filtering operation per row of the images
	- we have modified the rest of the code to run on the GPU, except for this kernel
    - implement the kernel, parallelizing loop over 'y' on the GPU, and finish the kernel launch code
	- start from a minimal-modifications approach
	
  float a = exp(-sqrt(2.0f) / sigma); // feedback coefficient
  for(int y = 0; y < h; y++) { 
    //filter left to right
    for(int x = 0; x < w-1; x++) {
      float coef = pow(a,dHoriz[y*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(y*w + x+1)*numCh + c] += coef * (imData[(y*w + x)*numCh + c] - imData[(y*w + x+1)*numCh + c]);
      }
    }
    //filter right to left
    for(int x = w-2; x >= 0; x--) {
      float coef = pow(a,dHoriz[y*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(y*w + x)*numCh + c] += coef * (imData[(y*w + x+1)*numCh + c] - imData[(y*w + x)*numCh + c]);
      }
    }
  } 
  */

  // write your own code here...
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  float a = exp(-sqrt(2.0f) / sigma);
  if(index < h)
  {
    //filter left to right
    for(int x = 0; x < w-1; x++) {
      float coef = pow(a,dHoriz[index*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(index*w + x+1)*numCh + c] += coef * (imData[(index*w + x)*numCh + c] - imData[(index*w + x+1)*numCh + c]);
      }
    }
    //filter right to left
    for(int x = w-2; x >= 0; x--) {
      float coef = pow(a,dHoriz[index*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(index*w + x)*numCh + c] += coef * (imData[(index*w + x+1)*numCh + c] - imData[(index*w + x)*numCh + c]);
      }
    }
  }
}

void recursiveFiltH(const float * dHoriz, float * imData, int h, int w, float sigma) {
  // the pointers passed as arguments to this function are 
  // already pointing to the data on the GPU
  dim3 threads(128); // MODIFY THIS LINE, use 128 threads per block
  dim3 grid(ceil(h/128)); // MODIFY THIS LINE, create the required number of blocks
  recursiveFiltHKernel<<<grid, threads>>>(dHoriz, imData, h, w, sigma);
}


GpuMat *distH_d = 0, *distVT_d = 0, *img_d = 0, *imRes_d = 0, *imResT_d = 0;

void computeDomainTransformFiltering(Mat &img, Mat &imRes, float sigma_s, float sigma_r, int numIter) {

  Size s = img.size();
  Mat imgT(img.t());
  Mat distH(s.height, s.width-1, CV_32FC3);
  Mat distVT(s.width, s.height-1, CV_32FC3);

  //allocate memory on GPU
  if(distH_d == 0) {
	//allocate memory on first run
    distH_d = new GpuMat();
    distVT_d = new GpuMat();
    img_d = new GpuMat();
    imRes_d = new GpuMat();
    imResT_d = new GpuMat();
    createContinuous(distH.size(), CV_32FC3, *distH_d);
    createContinuous(distVT.size(), CV_32FC3, *distVT_d);
    createContinuous(img.size(), CV_32FC3, *img_d);
    createContinuous(imRes.size(), CV_32FC3, *imRes_d);
    createContinuous(imRes.t().size(), CV_32FC3, *imResT_d);
  }

  // 1)  generate distance images
  computeDistanceH((float*)distH.data, (float*)img.data, s.height, s.width, sigma_s, sigma_r);
  computeDistanceH((float*)distVT.data, (float*)imgT.data, s.width, s.height, sigma_s, sigma_r);

  //copy data to GPU
  distH_d->upload(distH);
  distVT_d->upload(distVT);
  img_d->upload(img);
  imRes_d->upload(imRes);

  // 2) iterate filter
  for(int i = 0; i < numIter; i++) {
    // determine sigma (eq. 14)
    float sigma_H_i = sigma_s * sqrt(3) * pow(2, numIter - (i + 1)) / sqrt(pow(4, numIter) - 1);

    // filter horizontal
    recursiveFiltH((float*)distH_d->data, (float *)imRes_d->data, s.height, s.width, sigma_H_i);
    // filter vertical
    transpose((float *)imRes_d->data, (float *)imResT_d->data, s.height, s.width);
    recursiveFiltH((float *)distVT_d->data, (float *)imResT_d->data, s.width, s.height, sigma_H_i);
    transpose((float *)imResT_d->data, (float *)imRes_d->data, s.width, s.height);
  }
  hipDeviceSynchronize();
  imRes_d->download(imRes);
}
