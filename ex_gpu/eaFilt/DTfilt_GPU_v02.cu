#include "hip/hip_runtime.h"
__global__
void transposeKernel(float * __restrict__ inp, float * __restrict__ outp, int inH, int inW) {
  // ANNOTATE FUNCTION/KERNEL ARGUMENTS 
  // MODIFY the content to parallelize over the channels in the innermost dimension (threadIdx.x)
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if(y >= inH) 
    return;
  //for(int y = 0; y < inH; y++) {
  for(int x = 0; x < inW; x++) {
      outp[(x*inH + y)*numCh + threadIdx.x] = inp[(y*inW + x)*numCh + threadIdx.x];
  }
  //}
}

void transpose(float *inp, float *outp, int inH, int inW) {
  dim3 threads(numCh, 128); // MODIFY the grid and thread blocks
  dim3 grid(1, (inH-1)/threads.y + 1);
  transposeKernel<<<grid, threads>>>(inp, outp, inH, inW);
}

__global__
void recursiveFiltHKernel(const float * __restrict__ dHoriz, float * __restrict__ imData, int h, int w, float sigma) {
  // ANNOTATE FUNCTION/KERNEL ARGUMENTS 
  // MODIFY the content to parallelize over the channels in the innermost dimension (threadIdx.x)
  float a = exp(-sqrt(2.0f) / sigma); // feedback coefficient
  
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if(y >= h)
    return;
  
  //filter left to right
  for(int x = 0; x < w-1; x++) {
    float coef = pow(a,dHoriz[y*(w-1) + x]);
    imData[(y*w + x+1)*numCh + threadIdx.x] += coef * (imData[(y*w + x)*numCh + threadIdx.x] - imData[(y*w + x+1)*numCh + threadIdx.x]);
  }
  //filter right to left
  for(int x = w-2; x >= 0; x--) {
    float coef = pow(a,dHoriz[y*(w-1) + x]);
    imData[(y*w + x)*numCh + threadIdx.x] += coef * (imData[(y*w + x+1)*numCh + threadIdx.x] - imData[(y*w + x)*numCh + threadIdx.x]);
  }
}

void recursiveFiltH(const float * dHoriz, float * imData, int h, int w, float sigma) {
  dim3 threads(numCh, 128); // MODIFY the grid and thread blocks
  dim3 grid(1, (h-1)/threads.y+1);
  recursiveFiltHKernel<<<grid, threads>>>(dHoriz, imData, h, w, sigma);
}


GpuMat *distH_d = 0, *distVT_d = 0, *img_d = 0, *imRes_d = 0, *imResT_d = 0;

void computeDomainTransformFiltering(Mat &img, Mat &imRes, float sigma_s, float sigma_r, int numIter) {

  Size s = img.size();
  Mat imgT(img.t());
  Mat distH(s.height, s.width-1, CV_32FC3);
  Mat distVT(s.width, s.height-1, CV_32FC3);

  //allocate memory on GPU
  if(distH_d == 0) {
	//allocate memory on first run
    distH_d = new GpuMat();
    distVT_d = new GpuMat();
    img_d = new GpuMat();
    imRes_d = new GpuMat();
    imResT_d = new GpuMat();
    createContinuous(distH.size(), CV_32FC3, *distH_d);
    createContinuous(distVT.size(), CV_32FC3, *distVT_d);
    createContinuous(img.size(), CV_32FC3, *img_d);
    createContinuous(imRes.size(), CV_32FC3, *imRes_d);
    createContinuous(imRes.t().size(), CV_32FC3, *imResT_d);
  }

  // 1)  generate distance images
  computeDistanceH((float*)distH.data, (float*)img.data, s.height, s.width, sigma_s, sigma_r);
  computeDistanceH((float*)distVT.data, (float*)imgT.data, s.width, s.height, sigma_s, sigma_r);

  //copy data to GPU
  distH_d->upload(distH);
  distVT_d->upload(distVT);
  img_d->upload(img);
  imRes_d->upload(imRes);

  // 2) iterate filter
  for(int i = 0; i < numIter; i++) {
    // determine sigma (eq. 14)
    float sigma_H_i = sigma_s * sqrt(3) * pow(2, numIter - (i + 1)) / sqrt(pow(4, numIter) - 1);

    // filter horizontal
    recursiveFiltH((float*)distH_d->data, (float *)imRes_d->data, s.height, s.width, sigma_H_i);
    // filter vertical
    transpose((float *)imRes_d->data, (float *)imResT_d->data, s.height, s.width);
    recursiveFiltH((float *)distVT_d->data, (float *)imResT_d->data, s.width, s.height, sigma_H_i);
    transpose((float *)imResT_d->data, (float *)imRes_d->data, s.width, s.height);
  }
  hipDeviceSynchronize();
  imRes_d->download(imRes);
}
