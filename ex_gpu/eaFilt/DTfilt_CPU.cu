#include "hip/hip_runtime.h"

void recursiveFiltH(const float *dHoriz, float *imData, int h, int w, float sigma) {
  float a = exp(-sqrt(2.0f) / sigma); // feedback coefficient
  for(int y = 0; y < h; y++) { 
    //filter left to right
    for(int x = 0; x < w-1; x++) {
      float coef = pow(a,dHoriz[y*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(y*w + x+1)*numCh + c] += coef * (imData[(y*w + x)*numCh + c] - imData[(y*w + x+1)*numCh + c]);
      }
    }
    //filter right to left
    for(int x = w-2; x >= 0; x--) {
      float coef = pow(a,dHoriz[y*(w-1) + x]);
      for(int c = 0; c < numCh; c++) {
        imData[(y*w + x)*numCh + c] += coef * (imData[(y*w + x+1)*numCh + c] - imData[(y*w + x)*numCh + c]);
      }
    }
  }
}

void computeDomainTransformFiltering(Mat &img, Mat &imRes, float sigma_s, float sigma_r, int numIter) {

  // 1) generate distance images
  Size s = img.size();
  Mat distH(s.height, s.width-1, CV_32FC3), distVT(s.width, s.height-1, CV_32FC3);
  Mat imgT(img.t());
  computeDistanceH((float*)distH.data, (float*)img.data, s.height, s.width, sigma_s, sigma_r);
  computeDistanceH((float*)distVT.data, (float*)imgT.data, s.width, s.height, sigma_s, sigma_r);
  
  // 2) iterate filter
  for(int i = 0; i < numIter; i++) {
    // determine sigma (eq. 14)
    float sigma_H_i = sigma_s * sqrt(3) * pow(2, numIter - (i + 1)) / sqrt(pow(4, numIter) - 1);
    // filter horizontal
    recursiveFiltH((float*)distH.data, (float *)imRes.data, s.height, s.width, sigma_H_i);
    // filter vertical
    imRes = Mat(imRes.t());
    recursiveFiltH((float *)distVT.data, (float *)imRes.data, s.width, s.height, sigma_H_i);
    imRes = Mat(imRes.t());
  }
}




