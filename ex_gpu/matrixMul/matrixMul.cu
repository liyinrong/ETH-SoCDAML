#include "hip/hip_runtime.h"
// include libraries
#include <stdlib.h>
#include <sys/resource.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include "hip/hip_runtime_api.h"

__global__ void matmatmulgpu( double *a, double *b, double *c, int lda )
{

  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

  // assign one thread per output - row major over whole matrix
  int row = /* WHAT INDEX? */ ;
  int col = /* WHAT INDEX? */ ;
  double sum = 0.0;
  for ( int k=0; k<lda; k++ ) {
    sum += a[row*lda+k] * b[k*lda+col];
  }
  c[tid] = sum;
  return;
  
}

int main () {

  // banner
  printf (  "\n     Matrix-Matrix Multiplication\n");
  printf (    "     CPU code Ported to GPU \n");

  // define parameters 
  int n = 1024;  // matrix dimension
  
  // allocate arrays
  double *a = (double *) malloc ( n*n*sizeof(double) );
  double *b = (double *) malloc ( n*n*sizeof(double) );
  double *c = (double *) malloc ( n*n*sizeof(double) );
  
  // initialize data
  for ( int row = 0; row<n; row++ ) {
    for ( int col = 0; col<n; col++ ) {
      // data is in row-major format
      a[row*n+col] = sin( 0.01*col ) + cos( 0.013*row );
      b[row*n+col] = sin( 0.017*col ) + cos( 0.03*row );
    }
  }
	  
  // record start time - use cuda events, accurate
  hipEvent_t t_start;
  hipEvent_t t_end;
  hipEventCreate (&t_start);
  hipEventCreate (&t_end);
  hipEventRecord (t_start,0);

  // PERFORM MULTIPLICATION
  {
  // allocate space on device;
  double * d_a;
  double * d_b;
  double * d_c;
  hipMalloc ( &d_a, n*n*sizeof(double ));
  hipMalloc ( &d_b, n*n*sizeof(double ));
  hipMalloc ( &d_c, n*n*sizeof(double ));

  // copy intput data to device
  hipMemcpy ( d_a, a, n*n*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy ( d_b, b, n*n*sizeof(double), hipMemcpyHostToDevice );

  // define grid and block sizes
  unsigned int kgrid = n;
  unsigned int kblock = n;    

  // call kernel
  matmatmulgpu <<<kgrid, kblock>>> ( d_a, d_b, d_c, n );

  // copy the result back to the host
  hipMemcpy ( c, d_c, n*n*sizeof(double), hipMemcpyDeviceToHost );

  }

  // record end time
  hipEventRecord (t_end,0);
  hipEventSynchronize(t_end);
  float et;
  hipEventElapsedTime (&et, t_start, t_end);

  // report results
  printf(  "     elapsedTime         = %4.4f seconds\n", (double)et/1000.);  // hipEventElapsedTime is in milliseconds
  printf(  "     gigaflops achieved  = %4.4f Gflops/s\n\n\n", 2.0e-6*n*n*n/et); // 2( * and + ) *n (inner dimension)*n^2(result size)/(time in ms.)

}
